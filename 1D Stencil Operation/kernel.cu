#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

#define N 10
#define BLOCK_SIZE 256

__global__ void stencilOperation(const int* input, int* output) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N) {
        // Perform the stencil operation
        output[index] = input[index - 1] + input[index] + input[index + 1];
    }
}

int main() {
    int input[N], output[N];
    int* d_input, * d_output;

    // Initialize input array
    for (int i = 0; i < N; ++i) {
        input[i] = i;
    }

    // Allocate device memory
    hipMalloc((void**)&d_input, N * sizeof(int));
    hipMalloc((void**)&d_output, N * sizeof(int));

    // Copy input array to device
    hipMemcpy(d_input, input, N * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel
    int numBlocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    stencilOperation << <numBlocks, BLOCK_SIZE >> > (d_input, d_output);

    // Copy result back to host
    hipMemcpy(output, d_output, N * sizeof(int), hipMemcpyDeviceToHost);

    // Print result
    printf("Input array: ");
    for (int i = 0; i < N; ++i) {
        printf("%d ", input[i]);
    }
    printf("\n");

    printf("Output array: ");
    for (int i = 0; i < N; ++i) {
        printf("%d ", output[i]);
    }
    printf("\n");

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);

    return 0;
}